#include "hip/hip_runtime.h"
//##############################################################################
//#                                                                            #
//#                          Virus Model                                       #
//#                                                                            #
//##############################################################################


// nvcc DataFitting.cu -o program.out && ./program.out 2.0 562800.0 12.0 6.0
// nvcc DataFitting.cu -o program.out && ./program.out 42.6 1760.0 49.0 6.6
// nvcc DataFitting.cu -o program.out && ./program.out 53.65351593854275 3004.978221023878 26.189933225521735 15.871340460973098 0.2516247774969964
// Working for all cell numbers?
// Proper parameters
// Two arrays for cells; one past, one present
// In the middle of modifing cerial code
/*  
    Using Camal case:
        C Functions start with lower case
        Variables start with upper case
*/

//beta = 2.0;
//rho = 562800.0;
//TauI = 12.0;
//TauE = 6.0;
//c = 0.13

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <ctime>
#include <math.h>
#include <random>
#include <iostream>
using namespace std;
#include <chrono>

#define PI 3.1415926535897932f

#define CODETESTINGCONDITIONS 0
#define RUNCPU 0

//Globals to setup the kernals
dim3 BlockConfig, GridConfig;

int INITIALVIRUS = 0;

//Simulation Parametersf
int CELL2CELL = 0;
int FREECELL = 1;
// 0.000625, 0.00128, 0.0025, 0.005, 0.01, 0.02, 0.04
float timestep = 0.005;    //Time step for model (No larger than 0.01 hour) 0.005 hr = 18 sec, (1/3600) hr = 1 sec
float endtime = (365)*24;   //Days in hours
int Save = (1/timestep); //the number of time the program saves to file, (1/timestep) results in 1 save every simulated hour
int NumberOfLayers = 430;//501535 cells //7 //20 //61 //193 //607 is a million hexigon in a circle
int StartRuns = 0;
int NumberOfRuns = 10;

//Physical Parameters
//float MOI = pow(10,0); //pow(10,-5) to 1
float beta = 0.0; //2.3*pow(10,-7); //Infiction rate, units: per hour
float rho = 0.0; //1920
float D = 60*60*6*powf(10,-12); //Diffusion rate at 37 degrees celsius unit: m^2/s //pow(6*10,-12) //3.96e-8
float c = 0.0; //Clearance rate, units: per hour 
float deltx = 25.0*pow(10,-6);
float deltxprime = deltx*2;
float Dtsx2 = D*timestep*pow(deltxprime,-2);

//Probability Constants
float TauI = 0.0;  //Avg time for infection
float TauE = 0.0;   //Avg time for eclipse
float ne = 30.0;    //Number of eclipse compartments?
float ni = 100.0;   //Number of infected compartments?
//float probi = 0.2;  //Probability per unit time of cell to cell infection (/hour)

//float beta = 2.0;
//float rho = 562800.0;
//float TauI = 12.0;
//float TauE = 6.0;

//Global Variables
char Path_to_Folder[100] = "";
char Directroy[100] = "";
char** LocationData;
char* cells;
char* cells_GPU;
float* ecl;
float* ecl_GPU;
float* inf;
float* inf_GPU;
float* vtemp;
float* vtemp_GPU;
float* th;
float* th_GPU;
float* ut;
float* ut_GPU;
float* EclipsePhaseLength;
float* EclipsePhaseLength_GPU;
float* InfectionPhaseLength;
float* InfectionPhaseLength_GPU;
int NumberOfCells;
int NumberDead;

int NumberDead1;
int NumberInfected1;
int NumberEclipse1;
int NumberHealthy1;
float AmountOfVirus;

hiprandState *state;

//Functions
//float Te(float TauE, float ne){
////    Picks a random number from the gamma distribution
////    The number is to be used as a time step in the Eclipse Time Matrix

//    return TauE;
//}

//float Ti(float TauI, float ni){
////    Picks a random number from the gamma distribution
////    The number is to be used as a time step in the Eclipse Time Matrix

//    return TauI;
//}

//float Te(float TauE, float ne){
////    Picks a random number from the gamma distribution
////    The number is to be used as a time step in the Eclipse Time Matrix
//    float num = sqrtf(ne/(ne+1));
//    float m = log(TauE*num);
//    float s=sqrtf(2*log(1/num));

//    random_device rd;
//    default_random_engine generator(rd());
//    lognormal_distribution<double> distribution(m,s);

//    return distribution(generator);
//}

float Te(float TauE, float ne){
//    Picks a random number from the gamma distribution
//    The number is to be used as a time step in the Eclipse Time Matrix
    random_device rd;
    default_random_engine generator(rd());
    gamma_distribution<double> distribution(ne, TauE/ne);

    return distribution(generator);
}

float Ti(float TauI, float ni){
//    Picks a random number from the gamma distribution
//    The number is to be used as a time step in the Infected Time Matrix
    random_device rd;
    default_random_engine generator(rd());
    gamma_distribution<double> distribution(ni, TauI/ni);

    return distribution(generator);
}

float PU1(){
//    Picks a random number from a uniform distribution
//    This probability
    random_device rd;
    default_random_engine generator(rd());
    uniform_real_distribution<double> distribution(0.0,1.0);

    return distribution(generator);
}

void creatingPathToFolderAndDirectory(int BigIndex, int NumberOfLayers, float MOI, float probi){
    char TransmissionType[10] = "";
    if (CELL2CELL == 1){
		if (FREECELL == 1){
            strcat(TransmissionType,"Both");
        }
        else {
            strcat(TransmissionType,"CELL2CELL");
        }
    }
    else if(CELL2CELL == 0){
	    if (FREECELL == 0){
            strcat(TransmissionType,"Neither");
        }
        else{
            strcat(TransmissionType,"FREECELL");
        }
    }
    
    char Buffer[10]; //Buffer String For Conversion To Char
    char TheCurrentTime[50];
    time_t RawTime = time(NULL);
    tm* SpecificMoment = localtime(&RawTime);

    strcpy(Path_to_Folder, "");
    strcpy(Directroy, "");
    
//    if(RUNCPU == 1){
    strcat(Path_to_Folder,"ViralModel/");
//    }
//    else{
//        strcat(Path_to_Folder,"/media/baylorfain/HDD/CurrentDir/");
//    }
//	strftime(TheCurrentTime, 50, "%m-%d/%I:%M", SpecificMoment);
    strftime(TheCurrentTime, 50, "%m-%d/", SpecificMoment);
    strcat(Path_to_Folder,TheCurrentTime);

    sprintf(Buffer,"%d",NumberOfLayers);
    strcat(Path_to_Folder,Buffer);
    strcat(Path_to_Folder,"_");
    sprintf(Buffer,"%d",BigIndex);
    strcat(Path_to_Folder,Buffer);
    strcat(Path_to_Folder,"-");
    strcat(Path_to_Folder,TransmissionType);
    strcat(Path_to_Folder,"_");

    sprintf(Buffer,"%.1f",beta);
    strcat(Path_to_Folder,Buffer);
    strcat(Path_to_Folder,"_");

    sprintf(Buffer,"%.1f",rho);
    strcat(Path_to_Folder,Buffer);
    strcat(Path_to_Folder,"_");

    sprintf(Buffer,"%.1f",TauI);
    strcat(Path_to_Folder,Buffer);
    strcat(Path_to_Folder,"_");

    sprintf(Buffer,"%.1f",TauE);
    strcat(Path_to_Folder,Buffer);
    
    strcat(Directroy,"mkdir -p ");
    strcat(Directroy,Path_to_Folder);
    int check = system(strdup(Directroy));
    if(check != 0){
        exit(0);
    }
}

void creatingCellLocations(){
    float SideLenght = (2.0/3.0);
    int RadiusScale = 0;
    for(int i=0; i<NumberOfLayers; i++){
        if(i == 0){
            RadiusScale = RadiusScale + 1;
        }
        else{
            if((i)%2 == 1){
                RadiusScale = RadiusScale + 1;
            }
            else{
                RadiusScale = RadiusScale + 2;
            }
        }
    }
    float RadiusOfCircle = SideLenght*RadiusScale;

    int count = 0;
    for(int i=0; i<NumberOfLayers; i++){
        count = count + i;
    }
    int NumberOfHexagons=(count)*6+1;

    float** coord;
    int n = NumberOfHexagons;
    int m = 3;
    coord = (float**) calloc(n,sizeof(float*));  
    for (int i = 0; i < n; i++){
       coord[i] = (float*) calloc(m,sizeof(float));
    }

    float** percyclecoord;
    n = NumberOfHexagons;
    m = 3;
    percyclecoord = (float**) calloc(n,sizeof(float*));  
    for (int i = 0; i < n; i++){
       percyclecoord[i] = (float*) calloc(m,sizeof(float));  
    }

    int temp;
    for(int j=0; j<NumberOfLayers; j++){
        for(int i=0; i<(2*j); i++){
            if(i < j){
                temp = i;
            }
            percyclecoord[i+(j-1)*j+1][0] =  -temp-1;
            percyclecoord[i+(j-1)*j+1][1] =   temp+j-i;
            percyclecoord[i+(j-1)*j+1][2] =  -j+1+i;
            
        }
    }
    float c0[3] = {percyclecoord[0][0], percyclecoord[0][1], percyclecoord[0][2]};
    coord[0][2] = c0[2];
    coord[0][1] = c0[1];
    coord[0][0] = c0[0];

    count = 0;
    for(int j=0; j<(NumberOfHexagons/3); j++){
        for(int i=0; i<3; i++){
            coord[(i+0)%3+3*j+1][2] = percyclecoord[j+1][i]+c0[i];
            coord[(i+1)%3+3*j+1][1] = percyclecoord[j+1][i]+c0[i];
            coord[(i+2)%3+3*j+1][0] = percyclecoord[j+1][i]+c0[i];
        }
    }

    float hi = coord[0][0];
    float vi = coord[0][2];
    float xmin = INFINITY;
    float xcoord;
    float ycoord;
    double dist;
    for(int i=0; i<NumberOfHexagons; i++){
        xcoord = coord[i][0];
        if(coord[i][0] < xmin){
            xmin = coord[i][0];
        }
        ycoord = (2.0*sin(PI*(60.0/180.0))*(coord[i][1]-coord[i][2])/3.0)+vi;
        dist = sqrtf(pow(double(xcoord-hi),2.0)+pow(double(ycoord-vi),2.0));
        if(dist >= RadiusOfCircle){
            coord[i][0] = 5000.0;
            coord[i][1] = 0.0;
            coord[i][2] = 0.0;
        }
    }

    n = ((2*NumberOfLayers)-1);
    m = ((2*NumberOfLayers)-1);
    LocationData = (char**) malloc(n*sizeof(char*));  
    for(int j=0; j<n; j++){ 
        LocationData[j] = (char*) malloc(m*sizeof(char));  
        for(int i=0; i<m; i++){
            LocationData[j][i] = 'o';
       }
    }
    
    NumberOfCells = 0;
    for(int i=0; i<NumberOfHexagons; i++){
        if(coord[i][0] != 5000.0){
            LocationData[int(coord[i][2])-int(xmin)][int(coord[i][0])-int(xmin)] = 'h';
            NumberOfCells = NumberOfCells + 1;
        }
    }

//    char File1[100] = "";
//    strcat(File1,Path_to_Folder);
//    strcat(File1,"/InitialCellLocations.txt");
//    FILE *outfile1 = fopen(File1,"a");
//    if (outfile1 == NULL){
//        printf("Error opening file1!\n");
//        exit(0);
//    }
//    
//    for(int i=0; i<((2*NumberOfLayers)-1); i++){
//        for(int j=0; j<((2*NumberOfLayers)-1); j++){
//            fprintf(outfile1,"%c,",LocationData[i][j]);
//        }
//            fprintf(outfile1,"\n");
//    }
//    fclose(outfile1);
    
    char File2[100] = "";
    strcat(File2,Path_to_Folder);
    strcat(File2,"/Parameters.txt");
    FILE *outfile2 = fopen(File2,"w");
    if (outfile2 == NULL){
        printf("Error opening file2!\n");
        exit(0);
    }
    fprintf(outfile2, "Hexagon Side Lenght = %f\n", SideLenght);
    fprintf(outfile2, "Number of Layers = %d\n", NumberOfLayers);
    fprintf(outfile2, "Radius of Circle = %f\n", RadiusOfCircle);
    fprintf(outfile2, "Number of Cells = %d\n", NumberOfCells);
    fclose(outfile2);
    
    for (int i = 0; i < NumberOfHexagons; i++){  
       free(coord[i]);  
    }     
    free(coord);
    
    for (int i = 0; i < NumberOfHexagons; i++){  
       free(percyclecoord[i]);  
    }     
    free(percyclecoord);
}

void allocateMemory(int Nx, int Ny){    
    //Produces a matrix for the cells
    cells = (char*) malloc(Nx*Ny*2*sizeof(char));
    //Produces a matrix that will track the amount virus above each cell
    vtemp = (float*) malloc(Nx*Ny*2*sizeof(float));
    //Produces a univeral time matrix (ut)
    ut = (float*) malloc(Nx*Ny*sizeof(float));
    
    //Produces a time matrix for after eclipse phase (e)
    ecl = (float*) malloc(Nx*Ny*sizeof(float));
    //Produces a time matrix for after infection phase (i)
    inf = (float*) malloc(Nx*Ny*sizeof(float));
    //Produces a time matrix hor healthy cells (t)
    th = (float*) malloc(Nx*Ny*sizeof(float));
    
    //Produces an array of eclipse phase durations for cells
    EclipsePhaseLength = (float*) malloc(Nx*Ny*sizeof(float));
    //Produces an array of infection phase durations for cells
    InfectionPhaseLength = (float*) malloc(Nx*Ny*sizeof(float));
    
}

void initailConditions(int Nx, int Ny){
    for(int j=0; j<Ny; j++){
        for(int i=0; i<Nx; i++){
            for(int k=0;k<2;k++){
                cells[i+Nx*j+Nx*Ny*k] = LocationData[i][j];
                vtemp[i+Nx*j+Nx*Ny*k] = 0.0;
            }
            ut[i+Nx*j] = 0.0;
            ecl[i+Nx*j] = 0.0;
            inf[i+Nx*j] = 0.0;
            th[i+Nx*j] = 0.0;
            EclipsePhaseLength[i+Nx*j] = Te(TauE,ne);
            InfectionPhaseLength[i+Nx*j]  = Ti(TauI,ni);
       }
    }
    if (INITIALVIRUS == 1){
        vtemp[(NumberOfLayers-1)+Nx*(NumberOfLayers-1)+Nx*Ny*0] = powf(10,9);
        vtemp[(NumberOfLayers-1)+Nx*(NumberOfLayers-1)+Nx*Ny*1] = powf(10,9);
    }
}

void infectANumberOfCellsRandomly(int Nx, int Ny, int Ni){
    if(CODETESTINGCONDITIONS == 1){
        cells[(NumberOfLayers-1)+Nx*(NumberOfLayers-1)+Nx*Ny*0] = 'i';
        cells[(NumberOfLayers-1)+Nx*(NumberOfLayers-1)+Nx*Ny*1] = 'i'; //Only the center cell
    }
    else {
        srand(time(NULL));
        int randx;
        int randy; 
        int NumberOfInfectedCellsCount = 0;
        while(NumberOfInfectedCellsCount < Ni){
            randx = (rand()%Nx);
            randy = (rand()%Ny);
            if((cells[randx+Nx*randy+Nx*Ny*0] != 'o') && (cells[randx+Nx*randy+Nx*Ny*0] == 'h')){
                cells[randx+Nx*randy+Nx*Ny*0] = 'e';
                cells[randx+Nx*randy+Nx*Ny*1] = 'e';
                ecl[randx+Nx*randy] = Te(TauE,ne);
                NumberOfInfectedCellsCount = NumberOfInfectedCellsCount + 1;
            }
        }
    }
}

void printToFileCellAndVirusInitial(int Nx, int Ny, int NumberOfLayers){
    char File3[100] = "";
    strcat(File3,Path_to_Folder);
    strcat(File3,"/cells_over_time.txt");
    FILE *outfile3 = fopen(File3,"w");
    if (outfile3 == NULL){
        printf("Error opening file3!\n");
        exit(0);
    }
    for(int i=0; i<((2*NumberOfLayers)-1); i++){
        for(int j=0; j<((2*NumberOfLayers)-1); j++){
            fprintf(outfile3,"%c,",LocationData[i][j]);
        }
            fprintf(outfile3,"\n");
    }
    fclose(outfile3);

    char File4[100] = "";
    strcat(File4,Path_to_Folder);
    strcat(File4,"/virus_over_time.txt");
    FILE *outfile4 = fopen(File4,"w");
    if (outfile4 == NULL){
        printf("Error opening file4!\n");
        exit(0);
    }
    for(int i=0; i<((2*NumberOfLayers)-1); i++){
        for(int j=0; j<((2*NumberOfLayers)-1); j++){
            fprintf(outfile4,"%f,",0.0);
        }
            fprintf(outfile4,"\n");
    }
    fclose(outfile4);
}

void printToFileCellAndVirusAnalysisInitial(int Nx, int Ny){
    NumberDead1 = 0;
    NumberInfected1 = 0;
    NumberEclipse1 = 0;
    NumberHealthy1 = 0;
    AmountOfVirus = 0.0;
    for(int j=0; j<Ny; j++){
        for(int i=0; i<Nx; i++){
            AmountOfVirus = AmountOfVirus + vtemp[i+Nx*j+Nx*Ny*0];
            
            if(cells[i+Nx*j+Nx*Ny*0] == 'd'){
                NumberDead1 = NumberDead1 + 1;
            }
            else if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                NumberInfected1 = NumberInfected1 + 1;
            }
            else if(cells[i+Nx*j+Nx*Ny*0] == 'e'){
                NumberEclipse1 = NumberEclipse1 +1;
            }
            else if(cells[i+Nx*j+Nx*Ny*0] == 'h'){
                NumberHealthy1 = NumberHealthy1 + 1;
            }
        }
    }
    
    char File9[100] = "";
    strcat(File9,Path_to_Folder);
    strcat(File9,"/PerTimeStep.txt");
    FILE *outfile9 = fopen(File9,"w");
    if (outfile9 == NULL){
        printf("Error opening file9!\n");
        exit(0);
    }
    
    fprintf(outfile9,"%0.0f, %d, %d, %d, %d, %f,", 0.0, NumberHealthy1, NumberEclipse1, NumberInfected1, NumberDead1, AmountOfVirus);
    fprintf(outfile9,"\n");

    fclose(outfile9);
}

void cerialViralTransmission(int Nx, int Ny, int cell2cell, int freecell, float probi){
        
        //The Healthy Cells' time
        int NumberHealthy = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'h'){
                    NumberHealthy = NumberHealthy + 1;
                }
            }
        }
        int** LocationHealthy;  
        LocationHealthy = (int**) malloc(NumberHealthy*sizeof(int*));  
        for (int i=0; i<NumberHealthy; i++){
           LocationHealthy[i] = (int*) malloc(2*sizeof(int));  
        }
        int Indexer = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'h'){
                    LocationHealthy[Indexer][0] = i;
                    LocationHealthy[Indexer][1] = j;
                    Indexer = Indexer + 1;
                }
            }
        }
        if(NumberHealthy != 0){
            int Row;
            int Column;
            for(int j=0; j<NumberHealthy; j++){
                Row = LocationHealthy[j][0];
                Column = LocationHealthy[j][1];
//                    Row is the row location of for a cell
//                    Column is the column location for a cell
                th[Row+Nx*Column] = th[Row+Nx*Column] + timestep;
//                    "th" is the time matrix for healthy cells
//                    "ts" is the time step for the model
            }
        }
        for (int i = 0; i < NumberHealthy; i++){  
           free(LocationHealthy[i]);  
        }     
        free(LocationHealthy);
        //Eclipse phase -> Infection
        int NumberEclipse = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'e'){
                    NumberEclipse = NumberEclipse + 1;
                }
            }
        }
        int** LocationEclipse;  
        LocationEclipse = (int**) malloc(NumberEclipse*sizeof(int*));  
        for (int i=0; i<NumberEclipse; i++){
           LocationEclipse[i] = (int*) malloc(2*sizeof(int));  
        }
        Indexer = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'e'){
                    LocationEclipse[Indexer][0] = i;
                    LocationEclipse[Indexer][1] = j;
                    Indexer = Indexer + 1;
                }
            }
        }

        if(NumberEclipse != 0){
            int Row;
            int Column;
            for(int j=0; j<NumberEclipse; j++){
                Row = LocationEclipse[j][0];
                Column = LocationEclipse[j][1];
//                    Row is the row location of for a cell
//                    Column is the column location for a cell
                if((ecl[Row+Nx*Column] + th[Row+Nx*Column]) < ut[Row+Nx*Column]){
                    cells[Row+Nx*Column+Nx*Ny*1] = 'i';
                    inf[Row+Nx*Column] = inf[Row+Nx*Column] + Ti(TauI, ni);
//                        "ecl" is the time matrix for after eclipse phase
//                        "th" is the time matrix for healthy cells
//                        "ut" is the univeral time matrix
//                        "cells" is the matrix of cells
//                        "inf" is the time matrix for after infection phase
                }
            }
        }
        
        for (int i = 0; i < NumberEclipse; i++){  
           free(LocationEclipse[i]);  
        }     
        free(LocationEclipse);
        
        //Infection spreads
        if(cell2cell == 1){
            int NumberInfected = 0;
            for(int j=0; j<Ny; j++){
                for(int i=0; i<Nx; i++){
                    if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                        NumberInfected = NumberInfected + 1;
                    }
                }
            }
            int** LocationInfected;  
            LocationInfected = (int**) malloc(NumberInfected*sizeof(int*));  
            for (int i=0; i<NumberInfected; i++){
               LocationInfected[i] = (int*) malloc(2*sizeof(int));  
            }
            int Indexer = 0;
            for(int j=0; j<Ny; j++){
                for(int i=0; i<Nx; i++){
                    if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                        LocationInfected[Indexer][0] = i;
                        LocationInfected[Indexer][1] = j;
                        Indexer = Indexer + 1;
                    }
                }
            }
                 
                 
            if(NumberInfected != 0){
                int Row;
                int Column;
                for(int j=0; j<NumberInfected; j++){
                    Row = LocationInfected[j][0];
                    Column = LocationInfected[j][1];
//                        #Row is the row location of for a cell
//                        #Column is the column location for a cell

                    int AboveRowExists = 1;
                    int LeftColumnExists = 1;
                    int BelowRowExists = 1;
                    int RightColumnExists = 1;   

                    int AboveRow = Row-1;   //row coordinate above cell
                    int LeftColumn = Column-1;   //column coordinate left of cell
                    int BelowRow = Row+1;   //row coordinate below cell
                    int RightColumn = Column+1;   //column coordinate right of cell
                    
//                    if the cell one row up doesn't exist, it's taken out of the equation
                    if(AboveRow < 0){         
                        AboveRowExists = 0;
                        AboveRow = 0;
                    }
//                    if the cell one column to the left doesn't exist, it's taken out of the equation
                    if(LeftColumn < 0){         
                        LeftColumnExists = 0;
                        LeftColumn = 0;
                    }
//                    if the cell one row down doesn't exist, it's taken out of the equation
                    if(BelowRow > Ny-1){
                        BelowRowExists = 0;
                        BelowRow = 0;
                    }
//                    if the cell one column to the right doesn't exist, it's taken out of the equation
                    if(RightColumn > Nx-1){
                        RightColumnExists = 0;
                        RightColumn = 0;
                    }

                    if(PU1()<probi*timestep){
                        if((LeftColumnExists == 1) && (cells[Row+Nx*LeftColumn+Nx*Ny*0] != 'o')){
                            if(cells[Row+Nx*LeftColumn+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*LeftColumn+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*LeftColumn] = Te(TauE,ne);
                            }
                        }

                        if((RightColumnExists == 1) && (cells[Row+Nx*RightColumn+Nx*Ny*0] != 'o')){
                            if(cells[Row+Nx*RightColumn+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*RightColumn+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*RightColumn] = Te(TauE,ne);
                            }
                        }

                        if((AboveRowExists == 1) && (cells[AboveRow+Nx*Column+Nx*Ny*0] != 'o')){
                            if(cells[AboveRow+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[AboveRow+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[AboveRow+Nx*Column] = Te(TauE,ne);
                            }
                        }

                        if((BelowRowExists == 1) && (cells[BelowRow+Nx*Column+Nx*Ny*0] != 'o')){
                            if(cells[BelowRow+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[BelowRow+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[BelowRow+Nx*Column] = Te(TauE,ne);
                            }
                        }

                        if((AboveRowExists == 1) && (RightColumnExists == 1) && (cells[AboveRow+Nx*RightColumn+Nx*Ny*0] != 'o')){
                            if(cells[AboveRow+Nx*RightColumn+Nx*Ny*0] == 'h'){
                                cells[AboveRow+Nx*RightColumn+Nx*Ny*1] = 'e';
                                ecl[AboveRow+Nx*RightColumn] = Te(TauE,ne);
                            }
                        }

                        if((BelowRowExists == 1) && (LeftColumnExists == 1) && (cells[BelowRow+Nx*LeftColumn+Nx*Ny*0] != 'o')){
                            if(cells[BelowRow+Nx*LeftColumn+Nx*Ny*0] == 'h'){
                                cells[BelowRow+Nx*LeftColumn+Nx*Ny*1] = 'e';
                                ecl[BelowRow+Nx*LeftColumn] = Te(TauE,ne);
                            }
                        }
                    }
                }
            }
            
            for (int i = 0; i < NumberInfected; i++){  
               free(LocationInfected[i]);  
            }     
            free(LocationInfected);
        }
        //Virus Spreads
        int NumberVirus = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] != 'o'){
                    NumberVirus = NumberVirus + 1;
                }
            }
        }
        int** LocationVirus;  
        LocationVirus = (int**) malloc(NumberVirus*sizeof(int*));  
        for (int i=0; i<NumberVirus; i++){
           LocationVirus[i] = (int*) malloc(2*sizeof(int));  
        }
        Indexer = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] != 'o'){
                    LocationVirus[Indexer][0] = i;
                    LocationVirus[Indexer][1] = j;
                    Indexer = Indexer + 1;
                }
            }
        }
            
        int Row;
        int Column;
        for(int j=0; j<NumberVirus; j++){
            Row = LocationVirus[j][0];
            Column = LocationVirus[j][1];
//                Row is the row location of for a cell
//                Column is the column location for a cell  

            int AboveRow = Row-1;   //row coordinate above cell
            int LeftColumn = Column-1;   //column coordinate left of cell
            int BelowRow = Row+1;   //row coordinate below cell
            int RightColumn = Column+1;   //column coordinate right of cell

            float rho2;
            if(cells[Row+Nx*Column+Nx*Ny*0] == 'i'){
                rho2 = rho;
            }
            else{
                rho2 = 0;
            }
//            where rho2 is a placeholder variable

//            if the cell one row up doesn't exist, it's taken out of the equation
            if(AboveRow < 0){
                AboveRow = Row;
            }
//            if the cell one column to the left doesn't exist, it's taken out of the equation
            if(LeftColumn < 0){
                LeftColumn = Column;
            }
//            if the cell one row down doesn't exist, it's taken out of the equation
            if(BelowRow > (Ny-1)){
                BelowRow = Row;
            }
//            if the cell one column to the right doesn't exist, it's taken out of the equation
            if(RightColumn > (Nx-1)){
                RightColumn = Column;
            }

            if(cells[AboveRow+Nx*Column+Nx*Ny*0] == 'o'){
                AboveRow = Row;
            }
            if(cells[AboveRow+Nx*RightColumn+Nx*Ny*0] == 'o'){
                AboveRow = Row;
                RightColumn = Column;
            }
            if(cells[Row+Nx*RightColumn+Nx*Ny*0] == 'o'){
                RightColumn = Column;
            }
            if(cells[BelowRow+Nx*Column+Nx*Ny*0] == 'o'){
                BelowRow = Row;
            }
            if(cells[Row+Nx*LeftColumn+Nx*Ny*0] == 'o'){
                LeftColumn = Column;
            }
            if(cells[BelowRow+Nx*LeftColumn+Nx*Ny*0] == 'o'){
                BelowRow = Row;
                LeftColumn = Column;
            }
            
            float NNN = (vtemp[AboveRow+Nx*Column+Nx*Ny*0] + vtemp[AboveRow+Nx*RightColumn+Nx*Ny*0] + vtemp[Row+Nx*RightColumn+Nx*Ny*0] + vtemp[BelowRow+Nx*Column+Nx*Ny*0] + vtemp[Row+Nx*LeftColumn+Nx*Ny*0] + vtemp[BelowRow+Nx*LeftColumn+Nx*Ny*0]);
        
            float VirusProduced = rho2*timestep;
            float VirusDecay = c*vtemp[Row+Nx*Column+Nx*Ny*0]*timestep;
            float VirusOut = 4.0*Dtsx2*vtemp[Row+Nx*Column+Nx*Ny*0];
            float VirusIn = 2.0*Dtsx2*NNN/3.0;

            vtemp[Row+Nx*Column+Nx*Ny*1] = vtemp[Row+Nx*Column+Nx*Ny*0] + VirusProduced - VirusOut + VirusIn - VirusDecay;
            if(vtemp[Row+Nx*Column+Nx*Ny*1] < pow(10.0,-10.0)){
                vtemp[Row+Nx*Column+Nx*Ny*1] = 0.0;
            }
                //probability of infect adaptive time step            
                if(freecell == 1){
                float probaility = PU1();
                float adaptedtimestep = timestep; //variable time step
                float adaptedtimestepcount = 1.0;
                float pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                while(pinfect > 1.0){
                    adaptedtimestep = adaptedtimestep/2.0;
                    pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                    adaptedtimestepcount = adaptedtimestepcount*2.0;
                }
                if(pinfect <= 1.0){
                    if(adaptedtimestepcount != 1.0){
                        pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                    }
                    while(adaptedtimestepcount != 1.0){
                        if(probaility < pinfect){
                            if(cells[Row+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*Column] = Te(TauE,ne);
                            }
                        }
                        adaptedtimestepcount = adaptedtimestepcount/2.0;
//                        adaptedtimestep = adaptedtimestep*2.0;
                        pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                    }
                    if(adaptedtimestepcount == 1.0){                    
                        vtemp[Row+Nx*Column+Nx*Ny*1] = vtemp[Row+Nx*Column+Nx*Ny*0] + VirusProduced - VirusOut + VirusIn - VirusDecay;
                        if(probaility < pinfect){
                            if(cells[Row+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*Column] = Te(TauE,ne);
                            }
                        }
                    }
                }
            }
        }
        
        for (int i = 0; i < NumberVirus; i++){  
           free(LocationVirus[i]);  
        }     
        free(LocationVirus);
         
        //kills cells
        int NumberInfected = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                    NumberInfected = NumberInfected + 1;
                }
            }
        }
        int** LocationInfected;  
        LocationInfected = (int**) malloc(NumberInfected*sizeof(int*));  
        for (int i=0; i<NumberInfected; i++){
           LocationInfected[i] = (int*) malloc(2*sizeof(int));  
        }
        Indexer = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                    LocationInfected[Indexer][0] = i;
                    LocationInfected[Indexer][1] = j;
                    Indexer = Indexer + 1;
                }
            }
        }
             
        if(NumberInfected != 0){  
            int Row;
            int Column;     
            for(int j=0; j<NumberInfected; j++){
                Row = LocationInfected[j][0];
                Column = LocationInfected[j][1];
//                    Row is the row location of for a cell
//                    Column is the column location for a cell
                if(ut[Row+Nx*Column] > (inf[Row+Nx*Column] + ecl[Row+Nx*Column] + th[Row+Nx*Column])){
                    cells[Row+Nx*Column+Nx*Ny*1] = 'd';
                    if(CODETESTINGCONDITIONS == 1){
                        cells[Row+Nx*Column+Nx*Ny*1] = 'i';
            }
//                        "ut" is the univeral time matrix
//                        "inf" is the time matrix for after infection phase
//                        "ecl" is the time matrix for after eclipse phase
//                        "th" is the time matrix for healthy cells
//                        "cells" is the matrix of cells
                }
            }
        }
        
        for (int i = 0; i < NumberInfected; i++){  
           free(LocationInfected[i]);  
        }     
        free(LocationInfected);
        
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                vtemp[i+Nx*j+Nx*Ny*0] = vtemp[i+Nx*j+Nx*Ny*1];
                cells[i+Nx*j+Nx*Ny*0] = cells[i+Nx*j+Nx*Ny*1];
            }
        }
        
        //The Universal Time for the cells is kept here (ut)
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                ut[i+Nx*j] = ut[i+Nx*j] + timestep;
            }
        }
}

void modifiedCerialViralTransmission(int Nx, int Ny, int cell2cell, int freecell, float probi){
        
        int NumberHealthy = 0;
        int NumberEclipse = 0;
        int NumberInfected = 0;
        int NumberVirus = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'h'){
                    NumberHealthy = NumberHealthy + 1;
                }
                if(cells[i+Nx*j+Nx*Ny*0] == 'e'){
                    NumberEclipse = NumberEclipse + 1;
                }
                if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                    NumberInfected = NumberInfected + 1;
                }
                if(cells[i+Nx*j+Nx*Ny*0] != 'o'){
                    NumberVirus = NumberVirus + 1;
                }
            }
        }
        
        int** LocationHealthy;  
        LocationHealthy = (int**) malloc(NumberHealthy*sizeof(int*));  
        for (int i=0; i<NumberHealthy; i++){
           LocationHealthy[i] = (int*) malloc(2*sizeof(int));  
        }
        int** LocationEclipse;  
        LocationEclipse = (int**) malloc(NumberEclipse*sizeof(int*));  
        for (int i=0; i<NumberEclipse; i++){
           LocationEclipse[i] = (int*) malloc(2*sizeof(int));  
        }
        int** LocationInfected;  
        LocationInfected = (int**) malloc(NumberInfected*sizeof(int*));  
        for (int i=0; i<NumberInfected; i++){
           LocationInfected[i] = (int*) malloc(2*sizeof(int));  
        }
        int** LocationVirus;  
        LocationVirus = (int**) malloc(NumberVirus*sizeof(int*));  
        for (int i=0; i<NumberVirus; i++){
           LocationVirus[i] = (int*) malloc(2*sizeof(int));  
        }
        
        int IndexerH = 0;
        int IndexerE = 0;
        int IndexerI = 0;
        int IndexerO = 0;
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                if(cells[i+Nx*j+Nx*Ny*0] == 'h'){
                    LocationHealthy[IndexerH][0] = i;
                    LocationHealthy[IndexerH][1] = j;
                    IndexerH = IndexerH + 1;
                }
                if(cells[i+Nx*j+Nx*Ny*0] == 'e'){
                    LocationEclipse[IndexerE][0] = i;
                    LocationEclipse[IndexerE][1] = j;
                    IndexerE = IndexerE + 1;
                }
                if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                    LocationInfected[IndexerI][0] = i;
                    LocationInfected[IndexerI][1] = j;
                    IndexerI = IndexerI + 1;
                }
                if(cells[i+Nx*j+Nx*Ny*0] != 'o'){
                    LocationVirus[IndexerO][0] = i;
                    LocationVirus[IndexerO][1] = j;
                    IndexerO = IndexerO + 1;
                }
            }
        }
        
        //The Healthy Cells' time
        if(NumberHealthy != 0){
            int Row;
            int Column;
            for(int j=0; j<NumberHealthy; j++){
                Row = LocationHealthy[j][0];
                Column = LocationHealthy[j][1];
//                    Row is the row location of for a cell
//                    Column is the column location for a cell
                th[Row+Nx*Column] = th[Row+Nx*Column] + timestep;
//                    "th" is the time matrix for healthy cells
//                    "ts" is the time step for the model
            }
        }
        
        //Eclipse phase -> Infection
        if(NumberEclipse != 0){
            int Row;
            int Column;
            for(int j=0; j<NumberEclipse; j++){
                Row = LocationEclipse[j][0];
                Column = LocationEclipse[j][1];
//                    Row is the row location of for a cell
//                    Column is the column location for a cell
                if((ecl[Row+Nx*Column] + th[Row+Nx*Column]) < ut[Row+Nx*Column]){
                    cells[Row+Nx*Column+Nx*Ny*1] = 'i';
                    inf[Row+Nx*Column] = inf[Row+Nx*Column] + Ti(TauI, ni);
//                        "ecl" is the time matrix for after eclipse phase
//                        "th" is the time matrix for healthy cells
//                        "ut" is the univeral time matrix
//                        "cells" is the matrix of cells
//                        "inf" is the time matrix for after infection phase
                }
            }
        }
        
        //Infection spreads
        if(cell2cell == 1){             
            if(NumberInfected != 0){
                int Row;
                int Column;
                for(int j=0; j<NumberInfected; j++){
                    Row = LocationInfected[j][0];
                    Column = LocationInfected[j][1];
//                        #Row is the row location of for a cell
//                        #Column is the column location for a cell

                    int AboveRowExists = 1;
                    int LeftColumnExists = 1;
                    int BelowRowExists = 1;
                    int RightColumnExists = 1;   

                    int AboveRow = Row-1;   //row coordinate above cell
                    int LeftColumn = Column-1;   //column coordinate left of cell
                    int BelowRow = Row+1;   //row coordinate below cell
                    int RightColumn = Column+1;   //column coordinate right of cell
                    
//                    if the cell one row up doesn't exist, it's taken out of the equation
                    if(AboveRow < 0){         
                        AboveRowExists = 0;
                        AboveRow = 0;
                    }
//                    if the cell one column to the left doesn't exist, it's taken out of the equation
                    if(LeftColumn < 0){         
                        LeftColumnExists = 0;
                        LeftColumn = 0;
                    }
//                    if the cell one row down doesn't exist, it's taken out of the equation
                    if(BelowRow > Ny-1){
                        BelowRowExists = 0;
                        BelowRow = 0;
                    }
//                    if the cell one column to the right doesn't exist, it's taken out of the equation
                    if(RightColumn > Nx-1){
                        RightColumnExists = 0;
                        RightColumn = 0;
                    }

                    if(PU1()<probi*timestep){
                        if((LeftColumnExists == 1) && (cells[Row+Nx*LeftColumn+Nx*Ny*0] != 'o')){
                            if(cells[Row+Nx*LeftColumn+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*LeftColumn+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*LeftColumn] = Te(TauE,ne);
                            }
                        }

                        if((RightColumnExists == 1) && (cells[Row+Nx*RightColumn+Nx*Ny*0] != 'o')){
                            if(cells[Row+Nx*RightColumn+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*RightColumn+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*RightColumn] = Te(TauE,ne);
                            }
                        }

                        if((AboveRowExists == 1) && (cells[AboveRow+Nx*Column+Nx*Ny*0] != 'o')){
                            if(cells[AboveRow+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[AboveRow+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[AboveRow+Nx*Column] = Te(TauE,ne);
                            }
                        }

                        if((BelowRowExists == 1) && (cells[BelowRow+Nx*Column+Nx*Ny*0] != 'o')){
                            if(cells[BelowRow+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[BelowRow+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[BelowRow+Nx*Column] = Te(TauE,ne);
                            }
                        }

                        if((AboveRowExists == 1) && (RightColumnExists == 1) && (cells[AboveRow+Nx*RightColumn+Nx*Ny*0] != 'o')){
                            if(cells[AboveRow+Nx*RightColumn+Nx*Ny*0] == 'h'){
                                cells[AboveRow+Nx*RightColumn+Nx*Ny*1] = 'e';
                                ecl[AboveRow+Nx*RightColumn] = Te(TauE,ne);
                            }
                        }

                        if((BelowRowExists == 1) && (LeftColumnExists == 1) && (cells[BelowRow+Nx*LeftColumn+Nx*Ny*0] != 'o')){
                            if(cells[BelowRow+Nx*LeftColumn+Nx*Ny*0] == 'h'){
                                cells[BelowRow+Nx*LeftColumn+Nx*Ny*1] = 'e';
                                ecl[BelowRow+Nx*LeftColumn] = Te(TauE,ne);
                            }
                        }
                    }
                }
            }
        }
        
        //Virus Spreads
        int Row;
        int Column;
        for(int j=0; j<NumberVirus; j++){
            Row = LocationVirus[j][0];
            Column = LocationVirus[j][1];
//                Row is the row location of for a cell
//                Column is the column location for a cell  

            int AboveRow = Row-1;   //row coordinate above cell
            int LeftColumn = Column-1;   //column coordinate left of cell
            int BelowRow = Row+1;   //row coordinate below cell
            int RightColumn = Column+1;   //column coordinate right of cell

            float rho2;
            if(cells[Row+Nx*Column+Nx*Ny*0] == 'i'){
                rho2 = rho;
            }
            else{
                rho2 = 0;
            }
//            where rho2 is a placeholder variable

//            if the cell one row up doesn't exist, it's taken out of the equation
            if(AboveRow < 0){
                AboveRow = Row;
            }
//            if the cell one column to the left doesn't exist, it's taken out of the equation
            if(LeftColumn < 0){
                LeftColumn = Column;
            }
//            if the cell one row down doesn't exist, it's taken out of the equation
            if(BelowRow > (Ny-1)){
                BelowRow = Row;
            }
//            if the cell one column to the right doesn't exist, it's taken out of the equation
            if(RightColumn > (Nx-1)){
                RightColumn = Column;
            }

            if(cells[AboveRow+Nx*Column+Nx*Ny*0] == 'o'){
                AboveRow = Row;
            }
            if(cells[AboveRow+Nx*RightColumn+Nx*Ny*0] == 'o'){
                AboveRow = Row;
                RightColumn = Column;
            }
            if(cells[Row+Nx*RightColumn+Nx*Ny*0] == 'o'){
                RightColumn = Column;
            }
            if(cells[BelowRow+Nx*Column+Nx*Ny*0] == 'o'){
                BelowRow = Row;
            }
            if(cells[Row+Nx*LeftColumn+Nx*Ny*0] == 'o'){
                LeftColumn = Column;
            }
            if(cells[BelowRow+Nx*LeftColumn+Nx*Ny*0] == 'o'){
                BelowRow = Row;
                LeftColumn = Column;
            }
            
            float NNN = (vtemp[AboveRow+Nx*Column+Nx*Ny*0] + vtemp[AboveRow+Nx*RightColumn+Nx*Ny*0] + vtemp[Row+Nx*RightColumn+Nx*Ny*0] + vtemp[BelowRow+Nx*Column+Nx*Ny*0] + vtemp[Row+Nx*LeftColumn+Nx*Ny*0] + vtemp[BelowRow+Nx*LeftColumn+Nx*Ny*0]);
        
            float VirusProduced = rho2*timestep;
            float VirusDecay = c*vtemp[Row+Nx*Column+Nx*Ny*0]*timestep;
            float VirusOut = 4.0*Dtsx2*vtemp[Row+Nx*Column+Nx*Ny*0];
            float VirusIn = 2.0*Dtsx2*NNN/3.0;

            vtemp[Row+Nx*Column+Nx*Ny*1] = vtemp[Row+Nx*Column+Nx*Ny*0] + VirusProduced - VirusOut + VirusIn - VirusDecay;
            if(vtemp[Row+Nx*Column+Nx*Ny*1] < pow(10.0,-10.0)){
                vtemp[Row+Nx*Column+Nx*Ny*1] = 0.0;
            }
                //probability of infect adaptive time step            
                if(freecell == 1){
                float probaility = PU1();
                float adaptedtimestep = timestep; //variable time step
                float adaptedtimestepcount = 1.0;
                float pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                while(pinfect > 1.0){
                    adaptedtimestep = adaptedtimestep/2.0;
                    pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                    adaptedtimestepcount = adaptedtimestepcount*2.0;
                }
                if(pinfect <= 1.0){
                    if(adaptedtimestepcount != 1.0){
                        pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                    }
                    while(adaptedtimestepcount != 1.0){
                        if(probaility < pinfect){
                            if(cells[Row+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*Column] = Te(TauE,ne);
                            }
                        }
                        adaptedtimestepcount = adaptedtimestepcount/2.0;
//                        adaptedtimestep = adaptedtimestep*2.0;
                        pinfect = vtemp[Row+Nx*Column+Nx*Ny*1]*beta*adaptedtimestep;
                    }
                    if(adaptedtimestepcount == 1.0){                    
                        vtemp[Row+Nx*Column+Nx*Ny*1] = vtemp[Row+Nx*Column+Nx*Ny*0] + VirusProduced - VirusOut + VirusIn - VirusDecay;
                        if(probaility < pinfect){
                            if(cells[Row+Nx*Column+Nx*Ny*0] == 'h'){
                                cells[Row+Nx*Column+Nx*Ny*1] = 'e';
                                ecl[Row+Nx*Column] = Te(TauE,ne);
                            }
                        }
                    }
                }
            }
        }
         
        //kills cells
        if(NumberInfected != 0){  
            int Row;
            int Column;     
            for(int j=0; j<NumberInfected; j++){
                Row = LocationInfected[j][0];
                Column = LocationInfected[j][1];
//                    Row is the row location of for a cell
//                    Column is the column location for a cell
                if(ut[Row+Nx*Column] > (inf[Row+Nx*Column] + ecl[Row+Nx*Column] + th[Row+Nx*Column])){
                    cells[Row+Nx*Column+Nx*Ny*1] = 'd';
                    if(CODETESTINGCONDITIONS == 1){
                        cells[Row+Nx*Column+Nx*Ny*1] = 'i';
            }
//                        "ut" is the univeral time matrix
//                        "inf" is the time matrix for after infection phase
//                        "ecl" is the time matrix for after eclipse phase
//                        "th" is the time matrix for healthy cells
//                        "cells" is the matrix of cells
                }
            }
        }
        
        for (int i = 0; i < NumberHealthy; i++){  
           free(LocationHealthy[i]);  
        }     
        free(LocationHealthy);
        for (int i = 0; i < NumberEclipse; i++){  
           free(LocationEclipse[i]);  
        }     
        free(LocationEclipse);
        for (int i = 0; i < NumberInfected; i++){  
           free(LocationInfected[i]);  
        }     
        free(LocationInfected);
        for (int i = 0; i < NumberVirus; i++){  
           free(LocationVirus[i]);  
        }     
        free(LocationVirus);
        
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                vtemp[i+Nx*j+Nx*Ny*0] = vtemp[i+Nx*j+Nx*Ny*1];
                cells[i+Nx*j+Nx*Ny*0] = cells[i+Nx*j+Nx*Ny*1];
            }
        }
        
        //The Universal Time for the cells is kept here (ut)
        for(int j=0; j<Ny; j++){
            for(int i=0; i<Nx; i++){
                ut[i+Nx*j] = ut[i+Nx*j] + timestep;
            }
        }
}

void printToFileCellAndVirus(int Nx, int Ny, int NumberOfLayers){
    char File5[100] = "";
    strcat(File5,Path_to_Folder);
    strcat(File5,"/cells_over_time.txt");
    FILE *outfile5 = fopen(File5,"a");
    if (outfile5 == NULL){
        printf("Error opening file5!\n");
        exit(0);
    }
    for(int i=0; i<((2*NumberOfLayers)-1); i++){
        for(int j=0; j<((2*NumberOfLayers)-1); j++){
            fprintf(outfile5,"%c,",cells[i+Nx*j+Nx*Ny*0]);
        }
            fprintf(outfile5,"\n");
    }
    fclose(outfile5);
   
    char File6[100] = "";
    strcat(File6,Path_to_Folder);
    strcat(File6,"/virus_over_time.txt");
    FILE *outfile6 = fopen(File6,"a");
    if (outfile6 == NULL){
        printf("Error opening file6!\n");
        exit(0);
    }
    for(int i=0; i<((2*NumberOfLayers)-1); i++){
        for(int j=0; j<((2*NumberOfLayers)-1); j++){
            fprintf(outfile6,"%f,",vtemp[i+Nx*j+Nx*Ny*1]);
        }
            fprintf(outfile6,"\n");
    }
    fclose(outfile6);
}

void printToFileCellAndVirusAnalysis(float timestep){
    char File8[100] = "";
    strcat(File8,Path_to_Folder);
    strcat(File8,"/PerTimeStep.txt");
    FILE *outfile8 = fopen(File8,"a");
    if (outfile8 == NULL){
        printf("Error opening file8!\n");
        exit(0);
    }
    
    fprintf(outfile8,"%0.0f, %d, %d, %d, %d, %f,", timestep+1, NumberHealthy1, NumberEclipse1, NumberInfected1, NumberDead1, AmountOfVirus);
    fprintf(outfile8,"\n");

    fclose(outfile8);
}

void createParameterFile(float timestep, int NumberofSavedTimeSteps, float endtime, float timestepcount, float AmountOfVirus, float beta, float rho, float TauI, float TauE, float D, float deltxprime, float c, float probi){
    char File7[100] = "";
    strcat(File7,Path_to_Folder);
    strcat(File7,"/Parameters.txt");
    FILE *outfile7 = fopen(File7,"a");
    if (outfile7 == NULL){
        printf("Error opening file7!\n");
        exit(0);
    }
    fprintf(outfile7, "Time Step = %f\n", timestep);
    fprintf(outfile7, "Number of Saved Time Steps = %d\n", NumberofSavedTimeSteps);
    fprintf(outfile7, "Initial End Time = %f\n", endtime);
    fprintf(outfile7, "Actual Hours Simulated = %f\n", timestepcount*timestep);
    fprintf(outfile7, "Final Amount of Virus = %f\n", AmountOfVirus);
    fprintf(outfile7, "beta = %f\n", beta);
    fprintf(outfile7, "rho = %f\n", rho);
    fprintf(outfile7, "TauI = %f\n", TauI);
    fprintf(outfile7, "TauE = %f\n", TauE);
    fprintf(outfile7, "log10(D) = %f\n", log10(D));
    fprintf(outfile7, "delta x = %f\n", deltxprime);
    fprintf(outfile7, "c = %f\n", c);
    fprintf(outfile7, "Probability of cell to cell infection: %f\n", probi);
    fclose(outfile7);
}

void freeMemory(){ 
    for(int i=0; i<((2*NumberOfLayers)-1); i++){
        free(LocationData[i]);
    }
    free(LocationData);  
    free(cells);   
    free(ecl);
    free(inf);  
    free(vtemp);
    free(th);
    free(ut);
    free(EclipsePhaseLength);
    free(InfectionPhaseLength);
    
    if(RUNCPU == 0){
	    hipFree(cells_GPU);
	    hipFree(ecl_GPU);
	    hipFree(inf_GPU);
	    hipFree(vtemp_GPU);
	    hipFree(th_GPU);
	    hipFree(ut_GPU);
	    hipFree(EclipsePhaseLength_GPU);
	    hipFree(InfectionPhaseLength_GPU);
	    hipFree(state);
    }
}

void errorCheck(const char *message){
  hipError_t  error;
  error = hipGetLastError();

  if(error != hipSuccess)
  {
    printf("\n CUDA ERROR: %s = %s\n", message, hipGetErrorString(error));
    exit(0);
  }
}

struct systemConstantsStruct
{
    float MOI;
    float beta;
    float rho;
    float D;
    float c;
    float deltx;
    float deltxprime;
    float Dtsx2;

    float TauI;
    float TauE;
    float ne;
    float ni;
    float probi;
    
    float timestep;
};

systemConstantsStruct SystemConstants;

void loadConstants(float MOI, float probi){
    SystemConstants.MOI = MOI;
    SystemConstants.beta = beta;
    SystemConstants.rho = rho;
    SystemConstants.D = D;
    SystemConstants.c = c;
    SystemConstants.deltx = deltx;
    SystemConstants.deltxprime = deltxprime;
    SystemConstants.Dtsx2 = Dtsx2;

    SystemConstants.TauI = TauI;
    SystemConstants.TauE = TauE;
    SystemConstants.ne = ne;
    SystemConstants.ni = ni;
    SystemConstants.probi = probi;
    
    SystemConstants.timestep = timestep;
}

void deviceSetupAndMemoryAllocation(int Nx, int Ny){

	BlockConfig.x = 16;
	BlockConfig.y = 16;
	BlockConfig.z = 1;
	
	GridConfig.x = (Nx-1)/BlockConfig.x + 1;
	GridConfig.y = (Ny-1)/BlockConfig.y + 1;
	GridConfig.z = 1;
	
	hipMalloc((void**)&cells_GPU, Nx*Ny*2*sizeof(char));
	errorCheck("hipMalloc cells Mem");
	hipMalloc((void**)&vtemp_GPU, Nx*Ny*2*sizeof(float));
	errorCheck("hipMalloc vtemp Mem");
	hipMalloc((void**)&ut_GPU, Nx*Ny*sizeof(float));
	errorCheck("hipMalloc ut Mem");
	
	hipMalloc((void**)&ecl_GPU, Nx*Ny*sizeof(float));
	errorCheck("hipMalloc ecl Mem");
	hipMalloc((void**)&inf_GPU, Nx*Ny*sizeof(float));
	errorCheck("hipMalloc inf Mem");
	hipMalloc((void**)&th_GPU, Nx*Ny*sizeof(float));
	errorCheck("hipMalloc th Mem");
	
	hipMalloc((void**)&EclipsePhaseLength_GPU, Nx*Ny*sizeof(float));
	errorCheck("hipMalloc EclipsePhaseLength Mem");
	hipMalloc((void**)&InfectionPhaseLength_GPU, Nx*Ny*sizeof(float));
	errorCheck("hipMalloc InfectionPhaseLength Mem");
}

__global__ void cuRand_Setup(hiprandState *state){
    int Row = threadIdx.x + blockIdx.x * blockDim.x;
    int Column =  threadIdx.y + blockIdx.y * blockDim.y;
    int offsetx = blockDim.x * gridDim.x;

    int id = Row+offsetx*Column;
    hiprand_init (clock64(), id, 0, state);

}

__device__ float PU_GPU(hiprandState *state){
//    Picks a random number from a uniform distribution

    float Random = hiprand_uniform(state);

    return Random;
}

__global__ void kernel(char *cells, float *vtemp, float *ut, float *ecl, float *inf, float *th,  float *epl, float *ipl, systemConstantsStruct constant, int cell2cell, int freecell, hiprandState *state, int NumberOfLayers, float probi){
   
    int Row = threadIdx.x + blockIdx.x * blockDim.x;
    int Column =  threadIdx.y + blockIdx.y * blockDim.y;
    
    int NX = (2*NumberOfLayers-1);
    int NY = (2*NumberOfLayers-1);
    int NXNY = NX*NY;

    if((cells[Row+NX*Column+NXNY*0] != 'o') && (Row+NX*Column+NXNY < 2*NXNY)){
        //Virus Spreads
        int AboveRow = Row-1;   //row coordinate above cell
        int LeftColumn = Column-1;   //column coordinate left of cell
        int BelowRow = Row+1;   //row coordinate below cell
        int RightColumn = Column+1;   //column coordinate right of cell

        float rho2;
        if(cells[Row+NX*Column+NXNY*0] == 'i'){
            rho2 = constant.rho;
        }
        else{
            rho2 = 0;
        }
//          where rho2 is a placeholder variable

//          if the cell one row up doesn't exist, it's taken out of the equation
        if(AboveRow < 0){
            AboveRow = Row;
        }
//          if the cell one column to the left doesn't exist, it's taken out of the equation
        if(LeftColumn < 0){
            LeftColumn = Column;
        }
//          if the cell one row down doesn't exist, it's taken out of the equation
        if(BelowRow > (NY-1)){
            BelowRow = Row;
        }
//          if the cell one column to the right doesn't exist, it's taken out of the equation
        if(RightColumn > (NX-1)){
            RightColumn = Column;
        }

        if(cells[AboveRow+NX*Column+NXNY*0] == 'o'){
            AboveRow = Row;
        }
        if(cells[AboveRow+NX*RightColumn+NXNY*0] == 'o'){
            AboveRow = Row;
            RightColumn = Column;
        }
        if(cells[Row+NX*RightColumn+NXNY*0] == 'o'){
            RightColumn = Column;
        }
        if(cells[BelowRow+NX*Column+NXNY*0] == 'o'){
            BelowRow = Row;
        }
        if(cells[Row+NX*LeftColumn+NXNY*0] == 'o'){
            LeftColumn = Column;
        }
        if(cells[BelowRow+NX*LeftColumn+NXNY*0] == 'o'){
            BelowRow = Row;
            LeftColumn = Column;
        }
        
        float NNN = (vtemp[AboveRow+NX*Column+NXNY*0] + vtemp[AboveRow+NX*RightColumn+NXNY*0] + vtemp[Row+NX*RightColumn+NXNY*0] + vtemp[BelowRow+NX*Column+NXNY*0] + vtemp[Row+NX*LeftColumn+NXNY*0] + vtemp[BelowRow+NX*LeftColumn+NXNY*0]);
        
        float VirusProduced = rho2*constant.timestep;
        float VirusDecay = constant.c*vtemp[Row+NX*Column+NXNY*0]*constant.timestep;
        float VirusOut = 4.0*constant.Dtsx2*vtemp[Row+NX*Column+NXNY*0];
        float VirusIn = 2.0*constant.Dtsx2*NNN/3.0;

        __syncthreads();
        
        vtemp[Row+NX*Column+NXNY*1] = vtemp[Row+NX*Column+NXNY*0] + VirusProduced - VirusOut + VirusIn - VirusDecay;
        if(vtemp[Row+NX*Column+NXNY*1] < pow(10.0,-10.0)){
            vtemp[Row+NX*Column+NXNY*1] = 0.0;
        }
        
        //The Cell behavior
        if(cells[Row+NX*Column+NXNY*0] == 'i'){
            // Infectied
            if(ut[Row+NX*Column] > (inf[Row+NX*Column] + ecl[Row+NX*Column] + th[Row+NX*Column])){
                cells[Row+NX*Column+NXNY*1] = 'd';
                if(CODETESTINGCONDITIONS == 1){
                    cells[Row+NX*Column+NXNY*1] = 'i';
                }
            }
        }
        else if(cells[Row+NX*Column+NXNY*0] == 'e'){
            // Eclipse
            if(ut[Row+NX*Column] > (ecl[Row+NX*Column] + th[Row+NX*Column])){
                cells[Row+NX*Column+NXNY*1] = 'i';
//                inf[Row+NX*Column] = inf[Row+NX*Column] + ipl[Row+NX*Column];
                inf[Row+NX*Column] = ipl[Row+NX*Column];
            }
        }
        else if(cells[Row+NX*Column+NXNY*0] == 'h'){
            // Healthy
//            th[Row+NX*Column] = th[Row+NX*Column] + constant.timestep; moved to end of "h"
            
            if(cell2cell == 1){  
                // Cell to cell transmission
                int AboveRow = Row-1;   //row coordinate above cell
                int LeftColumn = Column-1;   //column coordinate left of cell
                int BelowRow = Row+1;   //row coordinate below cell
                int RightColumn = Column+1;   //column coordinate right of cell
                
        //        if the cell one row up doesn't exist, it's taken out of the equation
                if(AboveRow < 0){         
                    AboveRow = 0;
                }
        //        if the cell one column to the left doesn't exist, it's taken out of the equation
                if(LeftColumn < 0){
                    LeftColumn = 0;
                }
        //        if the cell one row down doesn't exist, it's taken out of the equation
                if(BelowRow > NY-1){
                    BelowRow = 0;
                }
        //        if the cell one column to the right doesn't exist, it's taken out of the equation
                if(RightColumn > NX-1){
                    RightColumn = 0;
                }

                if(PU_GPU(state) < constant.probi*constant.timestep){
                    if(cells[Row+NX*LeftColumn+NXNY*0] == 'i'){
                        cells[Row+NX*Column+NXNY*1] = 'e';
                    }
                    if(cells[Row+NX*RightColumn+NXNY*0] == 'i'){
                        cells[Row+NX*Column+NXNY*1] = 'e';
                    }
                    if(cells[AboveRow+NX*Column+NXNY*0] == 'i'){
                        cells[Row+NX*Column+NXNY*1] = 'e';
                    }
                    if(cells[BelowRow+NX*Column+NXNY*0] == 'i'){
                        cells[Row+NX*Column+NXNY*1] = 'e';
                    }
                    if(cells[AboveRow+NX*RightColumn+NXNY*0] == 'i'){
                        cells[Row+NX*Column+NXNY*1] = 'e';
                    }
                    if(cells[BelowRow+NX*LeftColumn+NXNY*0] == 'i'){
                        cells[Row+NX*Column+NXNY*1] = 'e';
                    }
                    
                    ecl[Row+NX*Column] = epl[Row+NX*Column];
                }
            }
            
            if(freecell == 1){
                // Cell free transmission
                float probablity = PU_GPU(state);
                float adaptedtimestep = constant.timestep; //variable time step
                float adaptedtimestepcount = 1.0;
                float pinfect = vtemp[Row+NX*Column+NXNY*1]*constant.beta*adaptedtimestep;
                while(pinfect > 1.0){
                    adaptedtimestep = adaptedtimestep/2.0;
                    pinfect = vtemp[Row+NX*Column+NXNY*1]*constant.beta*adaptedtimestep;
                    adaptedtimestepcount = adaptedtimestepcount*2.0;
                }
                if(pinfect <= 1.0){
                    if(adaptedtimestepcount != 1.0){
                        pinfect = vtemp[Row+NX*Column+NXNY*1]*constant.beta*adaptedtimestep;
                    }
                    while(adaptedtimestepcount != 1.0){
                        if(probablity < pinfect){
                            cells[Row+NX*Column+NXNY*1] = 'e';
                            ecl[Row+NX*Column] = epl[Row+NX*Column];
                        }
                        adaptedtimestepcount = adaptedtimestepcount/2.0;
//                        adaptedtimestep = adaptedtimestep*2.0;
                        pinfect = vtemp[Row+NX*Column+NXNY*1]*constant.beta*adaptedtimestep;
                    }
                    if(adaptedtimestepcount == 1.0){                    
                        vtemp[Row+NX*Column+NXNY*1] = vtemp[Row+NX*Column+NXNY*0] + VirusProduced - VirusOut + VirusIn - VirusDecay;
                        if(probablity < pinfect){
                            cells[Row+NX*Column+NXNY*1] = 'e';
                            ecl[Row+NX*Column] = epl[Row+NX*Column];
                        }
                    }
                }
            }
            th[Row+NX*Column] = th[Row+NX*Column] + constant.timestep;
        }

        //The Universal Time for the cells is kept here (ut)
        ut[Row+NX*Column] = ut[Row+NX*Column] + constant.timestep;
        vtemp[Row+NX*Column+NXNY*0] = vtemp[Row+NX*Column+NXNY*1];
        cells[Row+NX*Column+NXNY*0] = cells[Row+NX*Column+NXNY*1];
    }
}

int main(int argc, char *argv[]){
    //Checks for Heisenberg status of viral diffusion
    if(D*timestep/pow(deltxprime,2.0) > 0.5){
        printf("%.1f",D*timestep/pow(deltxprime,2.0));
        printf("CHANGE PARAMETERS TO FIT DIFFUSION LIMITS. VALUE MUST BE UNDER 0.5. VALUE SHOWN ABOVE");
        exit(0);
    }
//Clear Terminal
system("clear");

//float MOI[6] = {powf(10,0), powf(10,-1), powf(10,-2), powf(10,-3), powf(10,-4), powf(10,-5)};
//float MOI[5] = {powf(10,-1), powf(10,-2), powf(10,-3), powf(10,-4), powf(10,-5)};
//float MOI[3] = {powf(10,-3), powf(10,-4), powf(10,-5)};
float MOI[1] = {powf(10,-4)};
//float MOI[1] = {powf(10,0)};

//float probi[4] = {0.2, 0.4, 0.6, 0.8};
float probi[1] = {0.2};

for(int q=0;q<(sizeof(MOI)/sizeof(MOI[0]));q++){
for(int k=0;k<(sizeof(probi)/sizeof(probi[0]));k++){

    beta = atof(argv[1]);
    rho = atof(argv[2]);
    TauI = atof(argv[3]);
    TauE = atof(argv[4]);
    c = atof(argv[5]);
    
    char File[50] = "Run:";
    strcat(File, argv[1]);
    strcat(File,",");
    strcat(File, argv[2]);
    strcat(File,",");
    strcat(File, argv[3]);
    strcat(File,",");
    strcat(File, argv[4]);
    strcat(File,",");
    strcat(File, argv[5]);
    strcat(File,".txt");
    FILE *outfile = fopen(File,"w");
    if (outfile == NULL){
        printf("Error opening file!\n");
        exit(0);
    }

    //Loop For The number Of Simulations To Run Per Setting
    for(int BigIndex=0;BigIndex<NumberOfRuns;BigIndex++){
//        auto start = chrono::high_resolution_clock::now();

//        printf("\nStarting run %d\n", (BigIndex+1));

        //Creating Save Path
        creatingPathToFolderAndDirectory(StartRuns+BigIndex, NumberOfLayers, MOI[q], probi[k]);
        //Creating placeholder variables for multipy runs
        int cell2cell = CELL2CELL;
        int freecell = FREECELL;

        //Building Cells
        creatingCellLocations();
        
        //Number of Cells
        //Number of initial infected cells
        int Ni = NumberOfCells*MOI[q]; if(Ni < 1){ printf("Use larger MOI"); exit(0);} 
        int Nx = (2*NumberOfLayers-1);      //Range of cells on x axis
        int Ny = (2*NumberOfLayers-1);      //Range of cells on y axis
        
        //Makeing empty matrices
        allocateMemory(Nx, Ny);
        
        //Initializing
        initailConditions(Nx, Ny);

        //Deletes files and initial with values
//        if(BigIndex == 0){
//            printToFileCellAndVirusInitial(Nx, Ny, NumberOfLayers);
//            printToFileCellAndVirusAnalysisInitial(Nx, Ny);     
//        }

//        printToFileCellAndVirusAnalysisInitial(Nx, Ny);
        fprintf(outfile,"%f,", 0.0);
        
        //Infects a random cell, now seen as (e)
        infectANumberOfCellsRandomly(Nx, Ny, Ni);

        if(RUNCPU == 0){
            hipMalloc((void**)&state, Nx*Ny*sizeof(int));
            errorCheck("hipMalloc Random Setup");
            cuRand_Setup<<<GridConfig,BlockConfig>>>(state);
            errorCheck("Random Setup");
        
            loadConstants(MOI[q], probi[k]);
            
            deviceSetupAndMemoryAllocation(Nx, Ny);

	        hipMemcpy( cells_GPU, cells, Nx*Ny*2*sizeof(char), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy cells HtoD");
	        hipMemcpy( vtemp_GPU, vtemp, Nx*Ny*2*sizeof(float), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy vtemp HtoD");
	        hipMemcpy( ut_GPU, ut, Nx*Ny*sizeof(float), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy ut HtoD");
	        
	        hipMemcpy( ecl_GPU, ecl, Nx*Ny*sizeof(float), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy ecl HtoD");
	        hipMemcpy( inf_GPU, inf, Nx*Ny*sizeof(float), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy inf HtoD");
	        hipMemcpy( th_GPU, th, Nx*Ny*sizeof(float), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy th HtoD");
	        
	        hipMemcpy( EclipsePhaseLength_GPU, EclipsePhaseLength, Nx*Ny*sizeof(float), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy EclipsePhaseLength HtoD");
	        hipMemcpy( InfectionPhaseLength_GPU, InfectionPhaseLength, Nx*Ny*sizeof(float), hipMemcpyHostToDevice );
	        errorCheck("hipMemcpy InfectionPhaseLength HtoD");
        }
        
        //Runs simulation
        int NumberofTimeSteps = endtime/timestep;
        int NumberofSavedTimeSteps = NumberofTimeSteps/Save;
        int timestepcount = 0;    //equal to the number of ts elapsed
        while(timestepcount < (NumberofTimeSteps-1)){
            
            if(RUNCPU == 0){

                kernel<<<GridConfig,BlockConfig>>>(cells_GPU, vtemp_GPU, ut_GPU, ecl_GPU, inf_GPU, th_GPU, EclipsePhaseLength_GPU, InfectionPhaseLength_GPU, SystemConstants, cell2cell, freecell, state, NumberOfLayers, probi[k]);
            }
            else{
                //Cerial Viral Transmission
                cerialViralTransmission(Nx, Ny, cell2cell, freecell, probi[k]);
//                modifiedCerialViralTransmission(Nx, Ny, cell2cell, freecell, probi[k]);
            }
            
            if((timestepcount%Save) == 0){ 
                if(RUNCPU == 0){ 
                    hipMemcpy( cells, cells_GPU, Nx*Ny*2*sizeof(char), hipMemcpyDeviceToHost );
                    errorCheck("hipMemcpy cells DtoH");
                    hipMemcpy( vtemp, vtemp_GPU, Nx*Ny*2*sizeof(float), hipMemcpyDeviceToHost );
                    errorCheck("hipMemcpy vtemp DtoH");
                }

                //Analysisa dish
                NumberDead1 = 0;
                NumberInfected1 = 0;
                NumberEclipse1 = 0;
                NumberHealthy1 = 0;
                AmountOfVirus = 0.0;
                for(int j=0; j<Ny; j++){
                    for(int i=0; i<Nx; i++){
                        AmountOfVirus = AmountOfVirus + vtemp[i+Nx*j+Nx*Ny*0];
                        
                        if(cells[i+Nx*j+Nx*Ny*0] == 'd'){
                            NumberDead1 = NumberDead1 + 1;
                        }
                        else if(cells[i+Nx*j+Nx*Ny*0] == 'i'){
                            NumberInfected1 = NumberInfected1 + 1;
                        }
                        else if(cells[i+Nx*j+Nx*Ny*0] == 'e'){
                            NumberEclipse1 = NumberEclipse1 +1;
                        }
                        else if(cells[i+Nx*j+Nx*Ny*0] == 'h'){
                            NumberHealthy1 = NumberHealthy1 + 1;
                        }
                    }
                }
                        
                //Prints status of cells  virus
//                if(BigIndex == 0){
//                    printToFileCellAndVirus(Nx, Ny, NumberOfLayers);
//                    printToFileCellAndVirusAnalysis(timestepcount*timestep);
//                }
                
//                printToFileCellAndVirusAnalysis(timestepcount*timestep);
                fprintf(outfile,"%f,", AmountOfVirus);
            }
            
            //Number of days completed
//            if((timestepcount%(24*int(1/timestep))) == 0){
//                printf("%.0f Day\n",(timestepcount*timestep)/24);
//            }


//            if((NumberHealthy1 == 0)){
//                cell2cell = 0;
//                freecell = 0;
//            }
//            else{
//                cell2cell = CELL2CELL;
//                freecell = FREECELL;
//            }
//            //End Code if Virus has below 10
//            if((AmountOfVirus < pow(10,1.0)) && (NumberDead1 == NumberOfCells)){
//                break;
//            }

            if((NumberInfected1 == 0) && (NumberEclipse1 == 0)){
                cell2cell = 0;
                freecell = 0;
            }
            else{
                cell2cell = CELL2CELL;
                freecell = FREECELL;
            }
            //End Code if Virus has below 10
            if((AmountOfVirus < pow(10,5.0)) && (NumberInfected1 == 0) && (NumberEclipse1 == 0)){
                break;
            }
            
            timestepcount = timestepcount+1;
        }

        //Writes a file with all of our parameters/variables
        createParameterFile(timestep, NumberofSavedTimeSteps, endtime, timestepcount, AmountOfVirus, beta, rho, TauI, TauE, D, deltxprime, c, probi[k]);

//        printf("\nMOI(%.1f) probi(%.1f): %d of %d Runs Done\n", log10(MOI[q]), probi[k], (StartRuns+BigIndex+1), NumberOfRuns);

        freeMemory();
        
//        auto finish = std::chrono::high_resolution_clock::now();
//        chrono::duration<double> elapsed = finish - start;
//        cout << "Elapsed time: " << elapsed.count() << " s";
        
        fprintf(outfile,"\n");
    }
    fclose(outfile);
}
}
//printf("\nPROGRAM DONE\n");
}
